#include "hip/hip_runtime.h"
__global__
void matVecMulKnernel(float *M, float *V, float *P, int width){
    // A x A 矩阵乘上 A x 1向量
    int row = blockIdx.y * blockDim.y + threadIdx.y; // 处理哪一行
    if(row < width){
        // 有效范围内部
        float sum = 0;
        for(int i = 0;i < width; ++i){
            sum += M[row * width + i] * V[i]; // 计算dot product
        }
        P[row] = sum;
    }
}

void matVecMul(float *Vout, float *Min, float *Vin, int width){
    dim3 gridDim(1, width);
    dim3 blockDim(1, 1);
    float *Vout_d, *Min_d, *Vin_d;
    hipMalloc((void**)&Vout_d, sizeof(float) * width);
    hipMalloc((void**)&Min_d, sizeof(float) * width * width);
    hipMalloc((void**)&Vin_d, sizeof(float) * width);
    hipMemcpy(Min_d, Min, sizeof(float) * width * width, hipMemcpyHostToDevice);
    hipMemcpy(Vin_d, Vin, sizeof(float) * width, hipMemcpyHostToDevice);
    matVecMulKernel<<<gridDim, blockDim>>>(Min_d, Vin_d, Vout_d, width);
    hipMemcpy(Vout, Vout_d, sizeof(float) * width, hipMemcpyDeviceToHost);
    hipFree(Vout_d);
    hipFree(Min_d);
    hipFree(Vin_d);
}